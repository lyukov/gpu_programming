#include "CudaTimer.h"
#include ""

CudaTimer::CudaTimer() {
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent);
}

double CudaTimer::stop() {
    hipEventRecord(stopEvent);
    hipEventSynchronize(stopEvent);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
    return milliseconds / 1000.0;
}